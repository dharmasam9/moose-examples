#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#define CUDA_ERROR_CHECK
#define cusparseSafeCall( err ) __cusparseSafeCall( err, __FILE__, __LINE__ )

using namespace std;

inline void __cusparseSafeCall( hipsparseStatus_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( HIPSPARSE_STATUS_SUCCESS != err )
    {
        std::string errMsg;
        switch(err){
            case HIPSPARSE_STATUS_SUCCESS:
                errMsg =  "the operation completed successfully.";
                break;
            case HIPSPARSE_STATUS_NOT_INITIALIZED:
                errMsg =  "the library was not initialized.";
                break;
            case HIPSPARSE_STATUS_INVALID_VALUE:
                errMsg =  "invalid parameters were passed (mb,nnzb<=0).";
                break;
            case HIPSPARSE_STATUS_ARCH_MISMATCH:
                errMsg =  "the device only supports compute capability 2.0 and above.";
                break;
            case HIPSPARSE_STATUS_MAPPING_ERROR:
                errMsg =  "the texture binding failed.";
                break;
            case HIPSPARSE_STATUS_EXECUTION_FAILED:
                errMsg =  "the function failed to launch on the GPU.";
                break;
            case HIPSPARSE_STATUS_INTERNAL_ERROR:
                errMsg =  "an internal operation failed.";
                break;
            case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
                errMsg =  "the matrix type is not supported.";
                break;
            default:
                errMsg = "Unknown cusparse error";
                break;
        }

        fprintf( stderr, "cusparseSafeCall() failed at %s:%i : %s\n",
                 file, line, errMsg.c_str() );
        exit( -1 );
    }
#endif
 
    return;
}


int main(){
hipsparseHandle_t cusparse_handle=0;
hipsparseMatDescr_t cusparse_descr=0;

cusparseSafeCall(hipsparseCreate(&cusparse_handle));

return 0;
}

