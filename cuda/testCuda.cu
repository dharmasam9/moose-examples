#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdio>
#include <hip/hip_runtime.h>

#define CUDA_ERROR_CHECK
#define cudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

 
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
   
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
 
    return;
}

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  cudaSafeCall(hipMalloc(&d_x, N*sizeof(float))); 
  cudaSafeCall(hipMalloc(&d_y, N*sizeof(float)));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  cudaSafeCall(hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice));

// Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  cudaSafeCall(hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost));

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

}
